#include "hip/hip_runtime.h"
#include "kernel.cuh"



struct stackNode
{
	int index;
	int actualChild;
};

__device__ bool ray_triangle( const float4 V1,  // Triangle vertices
                           const float4 V2,
                           const float4 V3,
                           const float4 O,  //Ray origin
                           const float4 D,  //Ray direction
							float *t,
						   float &u, 
						   float &v)
{

	float4 e1, e2;  //Edge1, Edge2
	float4 P, Q, T;
	float det, inv_det;
	
	//Find vectors for two edges sharing V1
	SUB(e1, V2, V1);
	SUB(e2, V3, V1);
	//Begin calculating determinant - also used to calculate u parameter
	CROSS(P, D, e2);
	//if determinant is near zero, ray lies in plane of triangle
	det = DOT(e1, P);
	//NOT CULLING
	if(det > -EPSILON && det < EPSILON) return false;
	inv_det = 1.f / det;
 
	//calculate distance from V1 to ray origin
	SUB(T, O, V1);
 
	//Calculate u parameter and test bound
	u = DOT(T, P) * inv_det;
	//The intersection lies outside of the triangle
	if(u < 0.f - EPSILON || u > 1.f + EPSILON) return false;
 
	//Prepare to test v parameter
	CROSS(Q, T, e1);
 
	//Calculate V parameter and test bound
	v = DOT(D, Q) * inv_det;
	//The intersection lies outside of the triangle
	if(v < 0.f - EPSILON  || u + v  > 1.f + EPSILON) return false;
 
	*t = DOT(e2, Q) * inv_det;
 
	return *t > EPSILON; //ray intersection
}

__device__ bool ray_box(const float4 & O,  //Ray origin
						const float4 & D,  //Ray direction
						float * t,
						const float3 & amin,
						const float3 & amax)
{

	float md;
	float tmin = -1.0f *FLT_MAX ;
	float tmax = FLT_MAX;
	*t = FLT_MAX;
	float4 p;
	p.x = (amin.x + amax.x)/2.0f - O.x;
	p.y = (amin.y + amax.y)/2.0f - O.y;
	p.z = (amin.z + amax.z)/2.0f - O.z;

	float e,f,t1,t2,aux;

	for(int i=0;i<3;++i){
		md = (i==0)?abs(amax.x - amin.x)/2.0f:((i==1)?abs(amax.y - amin.y)/2.0f:abs(amax.z - amin.z)/2.0f);
		e = (i==0)?p.x:((i==1)?p.y:p.z);
		f = (i==0)?D.x:((i==1)?D.y:D.z);

		if ( abs(f) > 0.000001f ){
			t1 = (e + md)/f;
			t2 = (e - md)/f;

			if(t1 > t2){ aux = t1; t1 = t2; t2 = aux;}
			if(t1 > tmin) tmin = t1;
			if(t2 < tmax) tmax = t2;
			if(tmin > tmax) return false;
			if(tmax < 0) return false;	
		}else if(-e - md > 0 || -e + md < 0) return false;
	}

	*t = tmin;

	return true;
}

__device__ int octreeRayIntersection(	const float4 & O,  //Ray origin
										const float4 & D,  //Ray direction
										const Cell * octree,
										const uint3 * const id,
										const float4 * const pos,
										float *dist,
										float &u,
										float &v)
{
	int actual = 0;
	int init, num;
	uint3 idtri;
	stackNode Stack[MAXDEPTH + 2];
	float4 V0, V1, V2;
	float3 amin, amax;
	stackNode *Node;
	int idInter = -1, child_index;
	float t;
	float v1, u1;

	Stack[actual].index = 0;
	Stack[actual].actualChild = 0;
	actual++;

	while(actual > 0)
	{

		Node = &Stack[actual - 1];


		if(Node->actualChild >= octree[Node->index].numChilds)
		{
			--actual;
		}
		else if(octree[Node->index].type == LEAF)
		{

			init = octree[Node->index].firstChild;
			num = octree[Node->index].numChilds;

			for(int i = 0; i < num; ++i)
			{
				idtri = id[octree[init + i].firstChild];
				V0 = pos[idtri.x];
				V1 = pos[idtri.y];
				V2 = pos[idtri.z];

				t = FLT_MAX;
				if(ray_triangle(V0, V1, V2, O, D, &t, u1, v1) && t < *dist)
				{
					*dist = t;
					u = u1;
					v = v1;
					idInter = octree[init + i].firstChild; 
					//printf("Alguien aca");
				}
			}

			--actual;
		}
		else if(octree[Node->index].type == INTERNAL)
		{
			child_index = octree[Node->index].firstChild + Node->actualChild; 

			amin.x = octree[child_index].minBox.x;
			amin.y = octree[child_index].minBox.y;
			amin.z = octree[child_index].minBox.z;
				
			amax.x = octree[child_index].maxBox.x;
			amax.y = octree[child_index].maxBox.y;
			amax.z = octree[child_index].maxBox.z;

			if(ray_box(O, D, &t, amin, amax) 
				&& t < *dist){
				//Insert the new child in the stack
				Stack[actual].index = child_index;
				Stack[actual].actualChild = 0;
				++actual;
				if(Stack[actual].index == 9){ 
					actual = 3;
				}
			}

			++Node->actualChild;
		}
		else
		{
		/*	printf("AJA \n");
			for(unsigned int i =0; i< actual;++i){
				printf("%d %d\n", Stack[i].index, Stack[i].actualChild );
			}
			for(int k = 0;k<= Node->index; ++k)
				printf("%d %d %d\n",octree[k].type, INTERNAL, LEAF);*/
			printf("SHOULD NEVER COME HERE %d %d\n", Node->index, octree[Node->index].type);
		}
	}

	return idInter;
}



__global__ void kernelRC(uchar4 *buffer, const unsigned int width, const unsigned int height, 
						 const uint3 * const id, const float4 * const pos, const float4  * const normal, const float2 * const tex,  
						 const unsigned int num_vert, const unsigned int num_tri, const Options options, const Cell * const octree)
{
	/*unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int tpos = y * width + x;

	

	if(x < width && y < height)
	{

		buffer[tpos].x = 0;
		buffer[tpos].y = 0;
		buffer[tpos].z = 0;
		buffer[tpos].w = 0;

		float4 origin;
		float4 dir;
		float4 vaux;

		//dir(options.priX + x * options.incX, options.priY + y * options.incY, -1.0f ,0.0f)
		origin.x = 0.0f; origin.y = 0.0f; origin.z = 0; origin.w = 1;
		dir.x = options.priX + x * options.incX; dir.y = options.priY + y * options.incY; dir.z = -1; dir.w = 1;

		SUB(dir, dir, origin);

		vaux = origin;
		MULT(origin, options.modelView, vaux);

		vaux = dir;
		MULT(dir, options.modelView, vaux);


		unsigned int i=0;
		uint3 idtri;
		for(; i < num_tri; ++i)
		{
			//if(x>=300 && x<=600 && y >= 300 && y <= 600)
			//{
			idtri = id[i];
			float4 V0, V1, V2;

			V0 = pos[idtri.x];
			V1 = pos[idtri.y];
			V2 = pos[idtri.z];
			
			float t = FLT_MAX;
			if(ray_triangle( V0, V1, V2, origin, dir, &t))
			{
				buffer[tpos].x = 255;
				buffer[tpos].y = 255;
				buffer[tpos].z = 255;
				buffer[tpos].w = 255;
			}
			//}
		}
	}*/
	
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int tpos = y * width + x;
	float u, v;

	if(x < width && y < height)
	{

		int intersect = -1;
		float D = FLT_MAX;

		buffer[tpos].x = 0;
		buffer[tpos].y = 0;
		buffer[tpos].z = 0;
		buffer[tpos].w = 0;

		float4 origin;
		float4 dir;
		float4 vaux;
		//dir(options.priX + x * options.incX, options.priY + y * options.incY, -1.0f ,0.0f)
		origin.x = 0.0f; origin.y = 0.0f; origin.z = 0.0f; origin.w = 1.0f;
		dir.x = options.priX + x * options.incX; dir.y = options.priY + y * options.incY; dir.z = -1.0f; dir.w = 1.0f;

		SUB(dir, dir, origin);

		vaux = origin;
		MULT(origin, options.modelView, vaux);

		vaux = dir;
		MULT(dir, options.modelView, vaux);

		intersect = octreeRayIntersection(origin, dir, octree, id, pos, &D, u, v);
		if(intersect != -1)
		{

			uint3 idtri = id[intersect];
			float4 V0 = normal[idtri.x];
			float4 V1 = normal[idtri.y];
			float4 V2 = normal[idtri.z];
			if(u < 0.0f - EPSILON || u  > 1.0f + EPSILON  || v <0.0f - EPSILON  || v > 1.0f + EPSILON  ) printf("no deberia ocurrir %f %f %f \n",u, v, 1.0f - (u+v));
			BARI(buffer[tpos].x, 255.0f * V0.x, 255.0f * V1.x, 255.0f * V2.x, u, v);
			BARI(buffer[tpos].y, 255.0f * V0.y, 255.0f * V1.y, 255.0f * V2.y, u, v);
			BARI(buffer[tpos].z, 255.0f * V0.z, 255.0f * V1.z, 255.0f * V2.z, u, v);
			buffer[tpos].w = 255;
		}
	}
}



CUDAClass::CUDAClass()
{
	d_pos = NULL;
	d_normal = NULL;
	d_tex = NULL;
	d_id = NULL;
}

CUDAClass::~CUDAClass()
{
	checkCudaErrors(hipFree(d_pos));
	checkCudaErrors(hipFree(d_normal));
	checkCudaErrors(hipFree(d_tex));
	checkCudaErrors(hipFree(d_id));
	checkCudaErrors(hipFree(d_octree));

	hipDeviceReset();
}


void CUDAClass::cudaSetObject(const std::vector<CVertex> *ptr_puntos,const std::vector<CTriangle> *ptr_caras, const vector<Cell> *ptr_octree)
{
	float4 *h_pos = new float4[(*ptr_puntos).size()];
	float4 *h_normal = new float4[(*ptr_puntos).size()];
	float2 *h_tex = new float2[(*ptr_puntos).size()];
	uint3 *h_id = new uint3[(*ptr_caras).size()];

	num_vert = (*ptr_puntos).size();
	num_tri = (*ptr_caras).size();

	checkCudaErrors(hipMalloc((void**)&d_id,sizeof(uint3) * (*ptr_caras).size()));
	checkCudaErrors(hipMalloc((void**)&d_pos,sizeof(float4) * (*ptr_puntos).size()));
	checkCudaErrors(hipMalloc((void**)&d_normal,sizeof(float4) * (*ptr_puntos).size()));
	checkCudaErrors(hipMalloc((void**)&d_tex,sizeof(float2) * (*ptr_puntos).size()));
	checkCudaErrors(hipMalloc((void**)&d_octree, sizeof(Cell) * (*ptr_octree).size()));
	


	for(unsigned int i=0;i<(*ptr_caras).size();++i)
	{
		h_id[i].x = (*ptr_caras)[i].V0;
		h_id[i].y = (*ptr_caras)[i].V1;
		h_id[i].z = (*ptr_caras)[i].V2;
	}


	for(unsigned int i=0;i<(*ptr_puntos).size();++i)
	{
		h_pos[i].x = (*ptr_puntos)[i].v.x;
		h_pos[i].y = (*ptr_puntos)[i].v.y;
		h_pos[i].z = (*ptr_puntos)[i].v.z;
		h_pos[i].w = 1.0f;


		h_normal[i].x = (*ptr_puntos)[i].normal.x;
		h_normal[i].y = (*ptr_puntos)[i].normal.y;
		h_normal[i].z = (*ptr_puntos)[i].normal.z;
		h_normal[i].w = 0.0f;

		h_tex[i].x = (*ptr_puntos)[i].texture.x;
		h_tex[i].y = (*ptr_puntos)[i].texture.y;
	}

	checkCudaErrors(hipMemcpy(d_id,h_id, sizeof(uint3) * (*ptr_caras).size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_pos,h_pos, sizeof(float4) * (*ptr_puntos).size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_normal,h_normal, sizeof(float4) * (*ptr_puntos).size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tex,h_tex, sizeof(float2) * (*ptr_puntos).size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_octree, (*ptr_octree).data(), sizeof(Cell) * (*ptr_octree).size(), hipMemcpyHostToDevice));


	delete [] h_pos;
	delete [] h_normal;
	delete [] h_tex;
	delete [] h_id;

}


// Helper function for using CUDA to add vectors in parallel.
void CUDAClass::cudaRC(uchar4 *d_buffer, unsigned int width, unsigned int height, Options options)
{
	dim3 blockDim(16, 16, 1);
	dim3 gridDim((width + blockDim.x)/blockDim.x, (height + blockDim.y)/blockDim.y, 1);

	GpuTimer timer;
	timer.Start();
	kernelRC<<<gridDim, blockDim>>>(d_buffer, width, height, d_id, d_pos, d_normal, d_tex, num_vert, num_tri, options, d_octree);
	timer.Stop();

	printf("%f \n", timer.Elapsed());

	// Check for any errors launching the kernel
    checkCudaErrors(hipGetLastError());
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	checkCudaErrors(hipDeviceSynchronize());

	
}

